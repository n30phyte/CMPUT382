#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define BLOCK_SIZE 512 //TODO: You can change this

#define wbCheck(ans) gpuAssert((ans), __FILE__, __LINE__)

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void sort(int *d_deviceInput, int *d_deviceOutput, int numElements) {
    //TODO: Modify this to complete the functionality of the sort on the deivce

}

int main(int argc, char **argv) {
    wbArg_t args;
    int *hostInput;  // The input 1D list
    int *hostOutput; // The output list
    int *deviceInput;
    int *deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (int *) wbImport(wbArg_getInputFile(args, 0), &numElements, "integral_vector");
    hipHostAlloc(&hostOutput, numElements * sizeof(int), hipHostMallocDefault);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void **) &deviceInput, numElements * sizeof(int)));
    wbCheck(hipMalloc((void **) &deviceOutput, numElements * sizeof(int)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(int)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(int),
                       hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    wbTime_start(Compute, "Performing CUDA computation");
    sort(deviceInput, deviceOutput, numElements);
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                       hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    hipHostFree(hostOutput);

#if LAB_DEBUG
    system("pause");
#endif

    return 0;
}
