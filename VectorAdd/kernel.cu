#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void vecAdd(const float *in1, const float *in2, float *out, const int len) {
    unsigned int i = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}

void addVectors(std::vector<float> &input1, std::vector<float> &input2, std::vector<float> &output, int inputLength) {
    float *deviceInput1;
    float *deviceInput2;
    float *deviceOutput;

    hipMalloc((void **) &deviceInput1, inputLength * sizeof(float));
    hipMalloc((void **) &deviceInput2, inputLength * sizeof(float));
    hipMalloc((void **) &deviceOutput, inputLength * sizeof(float));

    hipMemcpy(deviceInput1, input1.data(), inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, input2.data(), inputLength * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int numBlocks = (inputLength + threadsPerBlock - 1) / threadsPerBlock;

    vecAdd <<< threadsPerBlock, numBlocks >>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
    hipDeviceSynchronize();

    hipMemcpy(output.data(), deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
}
