#include <hip/hip_runtime.h>
#include <>

#include "wb.h"

#include "exclusive_scan.h"

int main(int argc, char **argv) {
    wbArg_t args;
    float *hostInput;  // The input 1D list
    float *hostOutput; // The output list
    float *deviceInput;
    float *deviceOutput;
    float *auxScanBuffer;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hipHostAlloc(&hostOutput, numElements * sizeof(float),
                  hipHostMallocDefault);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ",
          numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void **) &deviceInput, numElements * sizeof(float));
    hipMalloc((void **) &deviceOutput, numElements * sizeof(float));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    hipMemset(deviceOutput, 0, numElements * sizeof(float));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    wbTime_start(Compute, "Performing CUDA computation");

    const int scanGridSize = (numElements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipMalloc((void **) &auxScanBuffer, scanGridSize * sizeof(float));
    recursiveScan(deviceInput, deviceOutput, numElements);
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    hipHostFree(hostOutput);

#if LAB_DEBUG
    system("pause");
#endif

    return 0;
}
