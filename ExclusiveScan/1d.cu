#include <hip/hip_runtime.h>
#include <>

#include "exclusive_scan.h"
#include "wb.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int main(int argc, char **argv) {
    wbArg_t args;
    float *hostInput;  // The input 1D list
    float *hostOutput; // The output list
    float *deviceInput;
    float *deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float *) wbImport(wbArg_getInputFile(args, 0), &numElements);
    hipHostAlloc(&hostOutput, numElements * sizeof(float),
                  hipHostMallocDefault);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ",
          numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void **) &deviceInput, numElements * sizeof(float)));
    wbCheck(hipMalloc((void **) &deviceOutput, numElements * sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                       hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    wbTime_start(Compute, "Performing CUDA computation");
    int gridSize = (numElements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    exclusiveScan<<<gridSize, BLOCK_SIZE>>>(deviceInput, deviceOutput, numElements);
    wbCheck(hipDeviceSynchronize());

    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                       hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    hipHostFree(hostOutput);

#if LAB_DEBUG
    system("pause");
#endif

    return 0;
}
