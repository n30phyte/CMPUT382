#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

#include "libwb/wb.h"

__device__ int binarySearch(const int value, const int *A, const int N) {
    int left = 0;
    int right = N - 1;

    int location = N;

    while (left <= right) {
        int middle = (left + right) / 2;

        if ((blockIdx.y == 0) ?     // Check if operating on A or B
            (A[middle] <= value) :  // If A, only return after value increases
            (A[middle] < value)) {  // If B, return as soon as you meet the value or it's larger

            left = middle + 1;
        } else {
            location = middle;
            right = middle - 1;
        }

    }
    return location;
}

__device__ int linearSearch(const int value, const int *A, const int N) {

    for (int i = 0; i < N; i++) {
        if ((blockIdx.y == 0) ?     // Check if operating on A or B
            (A[i] > value) :        // If A, only return after value increases
            (A[i] >= value)) {      // If B, return as soon as you meet the value or it's larger
            return i;
        }
    }

    return N;
}

__global__ void merge(int *C, const int *A, const int *B, const int N) {
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < N) {
        // Operate on a different array based on which block we're on.
        const int *source_array = (blockIdx.y == 0) ? A : B;
        const int *search_array = (blockIdx.y == 0) ? B : A;

        int i = linearSearch(source_array[threadId], search_array, N);
        C[threadId + i] = source_array[threadId];
    }
}

int main(int argc, char **argv) {
    wbArg_t args;
    int N;
    int *A;
    int *B;
    int *C;
    int *deviceA;
    int *deviceB;
    int *deviceC;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    A = (int *) wbImport(wbArg_getInputFile(args, 0), &N, NULL, "Integer");
    B = (int *) wbImport(wbArg_getInputFile(args, 1), &N, NULL, "Integer");
    C = (int *) malloc(2 * N * sizeof(int));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", N);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void **) &deviceA, N * sizeof(int));
    hipMalloc((void **) &deviceB, N * sizeof(int));
    hipMalloc((void **) &deviceC, 2 * N * sizeof(int));
    wbTime_stop(GPU, "Allocating GPU memory.");


    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceA, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B, N * sizeof(int), hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    // Perform on CUDA.
    const dim3 blockSize(threads, 1, 1);
    const dim3 gridSize(blocks, 2, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    merge <<<gridSize, blockSize>>>(deviceC, deviceA, deviceB, N);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(C, deviceC, 2 * N * sizeof(int), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, C, 2 * N);

    free(A);
    free(B);
    free(C);

#if LAB_DEBUG
    system("pause");
#endif

    return 0;
}
